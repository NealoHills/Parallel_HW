﻿#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

/*
Instead of calling getmax, call a
kernel getmaxcu() that outsources the job of finding the maximum number to the GPU. Let the main function allocate and populate the array in the host. Then you need to allocate memory in the device for that array and transfer the array from the host to the device, calculate the maximum in the device, then transfer that number back to the host.
*/

unsigned int getmax(unsigned int *, unsigned int);

__global__ void getmaxgpu(unsigned int *nums, unsigned int *global_max, unsigned int *size)
{
  __shared__ int block_max; // max of all threads in a block
  if (threadIdx.x == 0) { 
    block_max = 0;
  }
  __syncthreads();
  int index = threadIdx.x + (blockIdx.x * blockDim.x); // get thread index
  if(index < *size)
	  atomicMax(&block_max, nums[index]); // get max across all threads in the block

  __syncthreads();
  atomicMax(global_max, block_max); // get max across all blocks
}


int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    unsigned int * dev_numbers; // device copy of numbers
    unsigned int max = 0;
    unsigned int *d_max;
	unsigned int *d_size;
    int num_blocks;
    int num_threads = 256;
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);

    if (size < num_threads) {
      num_threads = size;
      num_blocks = size / num_threads;
	}
	else {
      num_blocks = size / num_threads;
	  if (size%num_threads) {
		  num_blocks += 1;
	  }
    }

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;

	//printf("The maximum number in the array (seq) is: %u\n", getmax(numbers, size));

    hipMalloc((void **)&dev_numbers, size * sizeof(unsigned int));
    hipMalloc((void **)&d_max, sizeof(unsigned int));
	hipMalloc((void **)&d_size, sizeof(unsigned int));
	hipMemcpy(d_max, &max, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_size, &size, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_numbers, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice);
	free(numbers);
	//cudaDeviceSynchronize();
    getmaxgpu<<<num_blocks, num_threads>>>(dev_numbers, d_max, d_size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
    }

    hipMemcpy(&max, d_max, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    hipFree(dev_numbers);
    hipFree(&d_max);
	hipFree(&d_size);

    hipDeviceSynchronize();
    
    printf("The maximum number in the array (gpu) is: %u\n", max);

    exit(0);
}

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/

unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );
}


